#include "hip/hip_runtime.h"
#include <cstddef>
#include <cstdint>
#include <cstdlib>
#include <filesystem>
#include <iostream>
#include <limits>
#include <optional>

#include "bam-api/bam_api.hpp"
#include "bam-api/bam_paired_reads.hpp"
#include "qmcp-solver/check_cuda_error.hpp"
#include "qmcp-solver/cuda_max_flow_solver.hpp"

__global__ void push_relabel_kernel(int* data) {
    // TODO(billyk):
}

__host__ void global_relabel(int* data) {
    // TODO(billyk):
}

qmcp::CudaMaxFlowSolver::CudaMaxFlowSolver() : is_data_loaded_(false) {}

qmcp::CudaMaxFlowSolver::CudaMaxFlowSolver(
    const std::filesystem::path& filepath)
    : is_data_loaded_(false) {
    import_data(filepath);
}

void qmcp::CudaMaxFlowSolver::import_data(
    const std::filesystem::path& filepath) {
    input_sequence_ = bam_api::BamApi::read_bam_soa(filepath);

    // Create max coverage function
    max_coverage_.resize(input_sequence_.ref_genome_length + 1, 0);
    for (bam_api::ReadIndex i = 0; i < input_sequence_.end_inds.size(); ++i) {
        for (bam_api::Index j = input_sequence_.start_inds[i];
             j <= input_sequence_.end_inds[i]; ++i) {
            ++max_coverage_[j + 1];
        }
    }

    is_data_loaded_ = true;
}
void qmcp::CudaMaxFlowSolver::add_edge(
    std::vector<std::vector<Node>>& neighbors_dict,
    std::vector<std::vector<EdgeDirection>>& edge_dir_dict,
    std::vector<std::vector<Capacity>>& residual_capacity_dict,
    std::vector<std::vector<uint32_t>>& inversed_edge_ind_dict, Node start,
    Node end, Capacity capacity) {
    size_t start_info_size = neighbors_dict[start].size();
    size_t end_info_size = neighbors_dict[end].size();

    neighbors_dict[start].push_back(end);
    edge_dir_dict[start].push_back(EdgeDirection::Forward);
    residual_capacity_dict[start].push_back(capacity);
    inversed_edge_ind_dict[start].push_back(end_info_size);

    neighbors_dict[end].push_back(start);
    edge_dir_dict[end].push_back(EdgeDirection::Backward);
    residual_capacity_dict[end].push_back(0);
    inversed_edge_ind_dict[end].push_back(start_info_size);
}

void qmcp::CudaMaxFlowSolver::create_graph(
    const bam_api::SOAPairedReads& sequence, uint32_t required_cover) {
    // Clear the graph data
    clear_graph();

    // Get genome length
    uint32_t n = sequence.ref_genome_length;

    // We first map 0, 1, 2, ... (n - 1) indices from the ref genome
    // to 1, 2, .... n. Then the graph is created in the following way, for read
    // (u, v), where u,v in (1, 2, ... n) create an edge (u - 1, v).
    //
    // The above procedure is equivalent of getting an original read (u, v),
    // where u,v in (0, 1, 2, ... n - 1) and createing an edge (u, v + 1).
    //
    // Additional nodes:
    // - source: n + 1,
    // - sink: n + 2,
    // - artificial node: 0.

    // Create source and sink
    Node source = n + 1;
    Node sink = n + 2;

    // Temporary dictionares with key=node
    std::vector<std::vector<Node>> neighbors_dict(n + 3);
    std::vector<std::vector<EdgeDirection>> edge_dir_dict(n + 3);
    std::vector<std::vector<Capacity>> residual_capacity_dict(n + 3);
    std::vector<std::vector<uint32_t>> inversed_edge_ind_dict(n + 3);

    // Add edges that are corresponding to the reads
    for (bam_api::ReadIndex i = 0; i < sequence.end_inds.size(); ++i) {
        Node u = sequence.start_inds[i];
        Node v = sequence.end_inds[i] + 1;

        // u --> v
        add_edge(neighbors_dict, edge_dir_dict, residual_capacity_dict,
                 inversed_edge_ind_dict, u, v, 1);
    }

    // Add returning edges
    for (Node i = 0; i < n; ++i) {
        // i + 1 --> i
        add_edge(neighbors_dict, edge_dir_dict, residual_capacity_dict,
                 inversed_edge_ind_dict, i + 1, i,
                 std::numeric_limits<Capacity>::max());
    }

    // Create demand func basing on the required cover
    std::vector<uint32_t> demand_func(n + 1, 0);
    for (bam_api::Index i = 0; i < n; ++i) {
        demand_func[i] = std::min(max_coverage_[i + 1], required_cover) -
                         std::min(max_coverage_[i], required_cover);
    }

    // Add edges for sink and source in order to simulate a circulation
    for (Node i = 0; i <= n; ++i) {
        if (demand_func[i] > 0) {
            // i --> sink
            add_edge(neighbors_dict, edge_dir_dict, residual_capacity_dict,
                     inversed_edge_ind_dict, i, sink, demand_func[i]);
        } else if (demand_func[i] < 0) {
            // source --> i
            add_edge(neighbors_dict, edge_dir_dict, residual_capacity_dict,
                     inversed_edge_ind_dict, source, i, -demand_func[i]);
        }
    }

    // Flatten the dictionaries and save them
    // Assumption: there are no isolated nodes
    neighbors_end_ind_.resize(n + 3, 0);
    neighbors_start_ind_.resize(n + 3, 0);

    uint32_t curr_ind = 0;

    for (bam_api::Index i = 0; i <= n + 2; ++i) {
        neighbors_start_ind_.push_back(curr_ind);
        neighbors_end_ind_.push_back(curr_ind + neighbors_dict[i].size());

        neighbors_.insert(neighbors_.end(), neighbors_dict[i].begin(),
                          neighbors_dict[i].end());

        residual_capacity_.insert(residual_capacity_.end(),
                                  residual_capacity_dict[i].begin(),
                                  residual_capacity_dict[i].end());

        edge_dir_.insert(edge_dir_.end(), edge_dir_dict[i].begin(),
                         edge_dir_dict[i].end());

        inversed_edge_ind_.insert(neighbors_end_ind_.end(),
                                  inversed_edge_ind_dict[i].begin(),
                                  inversed_edge_ind_dict[i].end());

        curr_ind += neighbors_dict[i].size() + 1;
    }

    // Prepare excess and label functions
    excess_func_.resize(n + 3, 0);
    label_func_.resize(n + 3, 0);
    label_func_[source] = n + 3;
    create_preflow();
}

void qmcp::CudaMaxFlowSolver::create_preflow() {
    // Get graph node count
    uint32_t n = label_func_.size();
    Node source = n - 2;

    // Create preflow: saturate all edges coming out of the source
    for (uint32_t i = neighbors_start_ind_[source];
         i <= neighbors_end_ind_[source]; ++i) {
        // We are in the source so every edge has forward direction
        // and checking the edge_dir is not requred

        // Get current neighbor
        Node curr_neighbor = neighbors_[i];
        Capacity curr_edge_capacity = residual_capacity_[i];

        // Get the inversed edge location
        uint32_t inversed_i =
            neighbors_start_ind_[curr_neighbor] + inversed_edge_ind_[i];

        // Saturate the edge
        residual_capacity_[inversed_i] = curr_edge_capacity;
        residual_capacity_[i] = 0;

        // Update the excess function
        excess_func_[curr_neighbor] = static_cast<Excess>(curr_edge_capacity);
        excess_func_[source] -= static_cast<Excess>(curr_edge_capacity);
    }
}

void qmcp::CudaMaxFlowSolver::clear_graph() {
    excess_func_.clear();
    label_func_.clear();
    neighbors_.clear();
    neighbors_start_ind_.clear();
    neighbors_end_ind_.clear();
    residual_capacity_.clear();
}

void qmcp::CudaMaxFlowSolver::solve(uint32_t required_cover) {
    if (!is_data_loaded_) {
        std::cerr << "Couldn't run solver: data has not been loaded.\n";
        std::exit(EXIT_FAILURE);
    }

    create_graph(input_sequence_, required_cover);

    Excess total_excess = 0;

    // Malloc and initialize CUDA memory
    int32_t* dev_label_func = nullptr;
    uint32_t* dev_excess_func = nullptr;
    uint32_t* dev_start_func_ind = nullptr;
    uint32_t* dev_end_func_ind = nullptr;
    Node* dev_neighbors = nullptr;
    Capacity* dev_residual_capacity = nullptr;
    EdgeDirection* dev_edge_dir = nullptr;

    CHECK_CUDA_ERROR(hipMalloc(reinterpret_cast<void**>(&dev_label_func),
                                label_func_.size() * sizeof(uint32_t)));
    CHECK_CUDA_ERROR(hipMemcpy(dev_label_func, label_func_.data(),
                                label_func_.size() * sizeof(uint32_t),
                                hipMemcpyHostToDevice));

    CHECK_CUDA_ERROR(hipMalloc(reinterpret_cast<void**>(&dev_excess_func),
                                excess_func_.size() * sizeof(int32_t)));
    CHECK_CUDA_ERROR(hipMemcpy(dev_excess_func, excess_func_.data(),
                                excess_func_.size() * sizeof(int32_t),
                                hipMemcpyHostToDevice));

    CHECK_CUDA_ERROR(
        hipMalloc(reinterpret_cast<void**>(&dev_start_func_ind),
                   neighbors_start_ind_.size() * sizeof(uint32_t)));
    CHECK_CUDA_ERROR(hipMemcpy(dev_start_func_ind, neighbors_start_ind_.data(),
                                neighbors_start_ind_.size() * sizeof(uint32_t),
                                hipMemcpyHostToDevice));

    CHECK_CUDA_ERROR(hipMalloc(reinterpret_cast<void**>(&dev_end_func_ind),
                                neighbors_end_ind_.size() * sizeof(uint32_t)));
    CHECK_CUDA_ERROR(hipMemcpy(dev_end_func_ind, neighbors_end_ind_.data(),
                                neighbors_end_ind_.size() * sizeof(uint32_t),
                                hipMemcpyHostToDevice));

    CHECK_CUDA_ERROR(hipMalloc(reinterpret_cast<void**>(&dev_neighbors),
                                neighbors_.size() * sizeof(Node)));
    CHECK_CUDA_ERROR(hipMemcpy(dev_neighbors, neighbors_.data(),
                                neighbors_.size() * sizeof(Node),
                                hipMemcpyHostToDevice));

    CHECK_CUDA_ERROR(
        hipMalloc(reinterpret_cast<void**>(&dev_residual_capacity),
                   residual_capacity_.size() * sizeof(Capacity)));
    CHECK_CUDA_ERROR(hipMemcpy(
        dev_residual_capacity, residual_capacity_.data(),
        residual_capacity_.size() * sizeof(Capacity), hipMemcpyHostToDevice));

    CHECK_CUDA_ERROR(hipMalloc(reinterpret_cast<void**>(&dev_edge_dir),
                                edge_dir_.size() * sizeof(EdgeDirection)));
    CHECK_CUDA_ERROR(hipMemcpy(dev_edge_dir, edge_dir_.data(),
                                edge_dir_.size() * sizeof(EdgeDirection),
                                hipMemcpyHostToDevice));

    // IMPLEMENTATION HERE

    CHECK_CUDA_ERROR(hipFree(dev_label_func));
    CHECK_CUDA_ERROR(hipFree(dev_excess_func));
    CHECK_CUDA_ERROR(hipFree(dev_start_func_ind));
    CHECK_CUDA_ERROR(hipFree(dev_end_func_ind));
    CHECK_CUDA_ERROR(hipFree(dev_neighbors));
    CHECK_CUDA_ERROR(hipFree(dev_residual_capacity));
    CHECK_CUDA_ERROR(hipFree(dev_edge_dir));
}
