#include <htslib/hts.h>
#include <stdio.h>
#include <cstdlib>

#include "app.hpp"
#include ""
#include "qmcp-solver/qmcp-solver.hpp"
#include "qmcp-solver/sequential_cost_scaling_network_solver.hpp"

int main(int argc, char** argv) {
    App app;

    try {
      app.Parse(argc, argv);
    } catch(const CLI::ParseError &e) {
      return app.Exit(e);
    }

    app.Solve();

    return EXIT_SUCCESS;
}
