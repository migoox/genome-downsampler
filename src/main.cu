#include "hip/hip_runtime.h"
#include <htslib/hts.h>
#include <stdio.h>

#include <chrono>
#include <filesystem>
#include <iostream>
#include <vector>

#include "bam-api/bam_api.hpp"
#include "hip/hip_runtime.h"
#include ""
#include "qmcp-solver/qmcp-solver.hpp"
#include "qmcp-solver/sequential_cost_scaling_network_solver.hpp"
hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size);

__global__ void addKernel(int* c, const int* a, const int* b) {
    int i = static_cast<int>(threadIdx.x);
    c[i] = a[i] + b[i];
}

int main() {
    // Bam api and qmcp solver test
    int M = 500;
    auto bam_path = std::filesystem::path(
        "/home/borys/Downloads/gpu-programming/data/ESIB_EQA_2023.SARS2.01/"
        "reads.bam");
    auto solver = qmcp::SequentialCostScalingNetworkSolver(M, bam_path);

    auto start = std::chrono::high_resolution_clock::now();
    solver.solve();
    auto stop = std::chrono::high_resolution_clock::now();
    auto solve_duration =
        std::chrono::duration_cast<std::chrono::seconds>(stop - start);

    std::cout << "SOLVE TOOK " << solve_duration.count() << "[seconds]"
              << std::endl;

    // Define some variables
    const int array_size = 5;
    const int a[array_size] = {1, 2, 3, 4, 5};
    const int b[array_size] = {10, 20, 30, 40, 50};
    int c[array_size] = {0};

    // Add vectors in parallel.
    hipError_t cuda_status = addWithCuda(c, a, b, array_size);
    if (cuda_status != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");  // NOLINT
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",  // NOLINT
           c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cuda_status = hipDeviceReset();
    if (cuda_status != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");  // NOLINT
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size) {
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;

    // Choose which GPU to run on, change this on a multi-GPU system.
    hipError_t cuda_status = hipSetDevice(0);
    if (cuda_status != hipSuccess) {
        fprintf(  // NOLINT
            stderr,
            "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;  // NOLINT
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cuda_status = hipMalloc((void**)&dev_c, size * sizeof(int));  // NOLINT
    if (cuda_status != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");  // NOLINT
        goto Error;                             // NOLINT
    }

    cuda_status = hipMalloc((void**)&dev_a, size * sizeof(int));  // NOLINT
    if (cuda_status != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");  // NOLINT
        goto Error;                             // NOLINT
    }

    cuda_status = hipMalloc((void**)&dev_b, size * sizeof(int));  // NOLINT
    if (cuda_status != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");  // NOLINT
        goto Error;                             // NOLINT
    }

    // Copy input vectors from host memory to GPU buffers.
    cuda_status =
        hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cuda_status != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");  // NOLINT
        goto Error;                             // NOLINT
    }

    cuda_status =
        hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cuda_status != hipSuccess) {
        fprintf(stderr,  // NOLINT
                "hipMemcpy failed!");
        goto Error;  // NOLINT
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cuda_status = hipGetLastError();
    if (cuda_status != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n",  // NOLINT
                hipGetErrorString(cuda_status));
        goto Error;  // NOLINT
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cuda_status = hipDeviceSynchronize();
    if (cuda_status != hipSuccess) {
        fprintf(stderr,  // NOLINT
                "hipDeviceSynchronize returned error code %d after launching "
                "addKernel!\n",
                cuda_status);
        goto Error;  // NOLINT
    }

    // Copy output vector from GPU buffer to host memory.
    cuda_status =
        hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cuda_status != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");  // NOLINT
        goto Error;                             // NOLINT
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cuda_status;
}
